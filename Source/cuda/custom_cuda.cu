#include "hip/hip_runtime.h"
#pragma once
#include "../../include/cuda/custom_cuda.cuh"
#include <thrust/count.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <opencv2/cudaarithm.hpp>
#include "../../include/allIncludes.h"

namespace custom_cuda
{
	__global__ void in_range_kernel(const cv::cuda::PtrStepSz<uchar3> src, cv::cuda::PtrStepSzb dst,
	                                int lbc0, int ubc0, int lbc1, int ubc1, int lbc2, int ubc2)
	{
		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x >= src.cols || y >= src.rows) return;

		uchar3 v = src(y, x);
		//if pixel >= threshold value, turn it white, else turn black
		if (v.x >= lbc0 && v.x <= ubc0 && v.y >= lbc1 && v.y <= ubc1 && v.z >= lbc2 && v.z <= ubc2)
			dst(y, x) = 255;
		else
			dst(y, x) = 0;
	}

	void in_range_gpu(cv::cuda::GpuMat& src, cv::Scalar& lowerb, cv::Scalar& upperb,
	                  cv::cuda::GpuMat& dst)
	{
		//Lower number = sharper result, but slower performance
		const int m = 16;
		int numRows = src.rows, numCols = src.cols;
		if (numRows == 0 || numCols == 0) return;
		// Attention! Cols Vs. Rows are reversed
		const dim3 gridSize(ceil((float)numCols / m), ceil((float)numRows / m), 1);
		const dim3 blockSize(m, m, 1);

		in_range_kernel<<<gridSize, blockSize>>>(src, dst, lowerb[0], upperb[0], lowerb[1], upperb[1],
		                                         lowerb[2], upperb[2]);
	}

	/*__global__ void get_perspective_transform_kernel(const cv::Point2f src[], const cv::Point2f dst[], int solve_method)
	{
		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;

		cv::cuda::GpuMat M(3, 3, CV_64F), X(8, 1, CV_64F, M.ptr());
		double a[8][8], b[8];
		cv::cuda::GpuMat A(8, 8, CV_64F, a), B(8, 1, CV_64F, b);

		for(int i = 0; i < 4; ++i)
		{
			a[i][0] = a[i + 4][3] = src[i].x;
			a[i][1] = a[i + 4][4] = src[i].y;
			a[i][2] = a[i + 4][5] = 1;
			a[i][3] = a[i][4] = a[i][5] =
			a[i + 4][0] = a[i + 4][1] = a[i + 4][2] = 0;
			a[i][6] = -src[i].x * dst[i].x;
			a[i][7] = -src[i].y * dst[i].x;
			a[i + 4][6] = -src[i].x * dst[i].y;
			a[i + 4][7] = -src[i].y * dst[i].y;
			b[i] = dst[i].x;
			b[i + 4] = dst[i].y;
		}
	}*/

	struct isNonZeroIndex
	{
		__host__ __device__ bool operator()(const int& idx)
		{
			return (idx != -1);
		}
	};

	__global__ void kernel_find_indices(const uint8_t *input, int width, int height, int step, float *indicesx, float *indicesy)
	{
		const int x = blockIdx.x * blockDim.x + threadIdx.x;
		const int y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x < width && y < height)
		{
			const int tidPixel = y * step + x;
			const int tidIndex = y * width + x;

			int value = input[tidPixel];
			if(value)
			{
				float X = float(x);
				float Y = float(y);
				indicesx[tidIndex] = X;
				indicesy[tidIndex] = Y;
			}
			else
			{
				indicesx[tidIndex] = -1;
				indicesy[tidIndex] = -1;
			}
		}
	}

	__global__ void processing_next(float* PointX_n, float* PointY_n, const float margin, float* left_n, float* right_n, const int N_n, float* LPoint_x, float* LPoint_y, float* RPoint_x, float* RPoint_y)
	{
		int i = threadIdx.x + blockIdx.x * blockDim.x;
		if (i < N_n)
		{
			float good_left_inds_n = ((PointX_n[i] > (left_n[0] * pow(float(PointY_n[i]), 2) + left_n[1] * PointY_n[i] + left_n[2] - margin)) & (PointX_n[i] < (left_n[0] * (pow(float(PointY_n[i]), 2)) + left_n[1] * PointY_n[i] + left_n[2] + margin)));
			float good_right_inds_n = ((PointX_n[i] > (right_n[0] * pow(float(PointY_n[i]), 2) + right_n[1] * PointY_n[i] + right_n[2] - margin)) & (PointX_n[i] < (right_n[0] * (pow(float(PointY_n[i]), 2)) + right_n[1] * PointY_n[i] + right_n[2] + margin)));

			if (good_left_inds_n != 0)
			{
				LPoint_x[i] = PointX_n[i];
				LPoint_y[i] = PointY_n[i];
			}
			else
			{
				LPoint_x[i] = -1;
				LPoint_y[i] = -1;
			}
			if (good_right_inds_n != 0)
			{
				RPoint_x[i] = PointX_n[i];
				RPoint_y[i] = PointY_n[i];
			}
			else
			{
				RPoint_x[i] = -1;
				RPoint_y[i] = -1;
			}
		}

	}

	void indices_point(cv::cuda::GpuMat& src, thrust::device_vector<float>& outx, thrust::device_vector<float>& outy)
	{
		int Array_Size = cv::cuda::countNonZero(src);
		thrust::device_vector<float>Point_x(src.rows * src.step);
		thrust::device_vector<float>Point_y(src.rows * src.step);
		uint8_t* imgPtr;
		hipMalloc((void**)&imgPtr, src.rows * src.step);
		hipMemcpyAsync(imgPtr, src.ptr<uint8_t>(), src.rows * src.step, hipMemcpyDeviceToDevice);
		dim3 block(16, 16);
		dim3 grid;
		grid.x = (src.cols + block.x - 1) / block.x;
		grid.y = (src.rows + block.y - 1) / block.y;
		kernel_find_indices << <grid, block >> > (imgPtr, int(src.cols), int(src.rows), int(src.step), thrust::raw_pointer_cast(Point_x.data()), thrust::raw_pointer_cast(Point_y.data()));
		hipDeviceSynchronize();
		thrust::copy_if(Point_x.begin(), Point_x.end(), outx.begin(), isNonZeroIndex());
		thrust::copy_if(Point_y.begin(), Point_y.end(), outy.begin(), isNonZeroIndex());
		hipFree(imgPtr);

	}

	void get_non_zero_pixels(cv::cuda::GpuMat& src, std::vector<float>& output_hx, std::vector<float>& output_hy)
	{

		int array_size = cv::cuda::countNonZero(src);
		thrust::device_vector<float>Point_X(array_size);
		thrust::device_vector<float>Point_Y(array_size);
		indices_point(src, Point_X, Point_Y);
		output_hx.resize(array_size);
		output_hy.resize(array_size);
		thrust::copy(Point_X.begin(), Point_X.end(), output_hx.begin());
		thrust::copy(Point_Y.begin(), Point_Y.end(), output_hy.begin());
	}

	void get_non_zero_pixels_next(cv::cuda::GpuMat& src, std::vector<float>& Loutput_hx, std::vector<float>& Loutput_hy, std::vector<float>& Routput_hx, std::vector<float>& Routput_hy)
	{

		std::vector<float>polyright_out_n;
		std::vector<float>polyleft_out_n;

		polyright_out_n = last_fit::polyfit_right;
		polyleft_out_n = last_fit::polyfit_left;
		size_t SIZE_T = 3 * sizeof(float);
		float* right_fit_last = (float*)malloc(SIZE_T);
		float* right_fit_last_d;
		hipMalloc(&right_fit_last_d, SIZE_T);
		float* left_fit_last = (float*)malloc(SIZE_T);
		float* left_fit_last_d;
		hipMalloc(&left_fit_last_d, SIZE_T);

		right_fit_last[2] = polyright_out_n[0];
		left_fit_last[2] = polyleft_out_n[0];
		right_fit_last[1] = polyright_out_n[1];
		left_fit_last[1] = polyleft_out_n[1];
		right_fit_last[0] = polyright_out_n[2];
		left_fit_last[0] = polyleft_out_n[2];

		hipMemcpy(right_fit_last_d, right_fit_last, SIZE_T, hipMemcpyHostToDevice);
		hipMemcpy(left_fit_last_d, left_fit_last, SIZE_T, hipMemcpyHostToDevice);


		const float margin = 10;
		const int Size_array = cv::cuda::countNonZero(src);
		thrust::device_vector<float>Point_X(Size_array);
		thrust::device_vector<float>Point_Y(Size_array);
		indices_point(src, Point_X, Point_Y);

		float* arrayx = thrust::raw_pointer_cast(&Point_X[0]);
		float* arrayy = thrust::raw_pointer_cast(&Point_Y[0]);

		thrust::device_vector<float>LPoint_x(Size_array);
		thrust::device_vector<float>LPoint_y(Size_array);
		thrust::device_vector<float>RPoint_x(Size_array);
		thrust::device_vector<float>RPoint_y(Size_array);

		processing_next << <Size_array, 1 >> > (arrayx, arrayy, margin, left_fit_last_d, right_fit_last_d, Size_array, thrust::raw_pointer_cast(LPoint_x.data()),
			thrust::raw_pointer_cast(LPoint_y.data()), thrust::raw_pointer_cast(RPoint_x.data()), thrust::raw_pointer_cast(RPoint_y.data()));
		hipDeviceSynchronize();

		int nonZeroCountL = int(thrust::count_if(LPoint_x.begin(), LPoint_x.end(), isNonZeroIndex()));
		int nonZeroCountR = int(thrust::count_if(RPoint_x.begin(), RPoint_x.end(), isNonZeroIndex()));

		thrust::device_vector<float>Loutx(nonZeroCountL);
		thrust::copy_if(LPoint_x.begin(), LPoint_x.end(), Loutx.begin(), isNonZeroIndex());
		Loutput_hx.resize(nonZeroCountL);
		thrust::copy(Loutx.begin(), Loutx.end(), Loutput_hx.begin());

		thrust::device_vector<float>Louty(nonZeroCountL);
		thrust::copy_if(LPoint_y.begin(), LPoint_y.end(), Louty.begin(), isNonZeroIndex());
		Loutput_hy.resize(nonZeroCountL);
		thrust::copy(Louty.begin(), Louty.end(), Loutput_hy.begin());

		thrust::device_vector<float>Routx(nonZeroCountR);
		thrust::copy_if(RPoint_x.begin(), RPoint_x.end(), Routx.begin(), isNonZeroIndex());
		Routput_hx.resize(nonZeroCountR);
		thrust::copy(Routx.begin(), Routx.end(), Routput_hx.begin());

		thrust::device_vector<float>Routy(nonZeroCountR);
		thrust::copy_if(RPoint_y.begin(), RPoint_y.end(), Routy.begin(), isNonZeroIndex());
		Routput_hy.resize(nonZeroCountR);
		thrust::copy(Routy.begin(), Routy.end(), Routput_hy.begin());

		hipFree(right_fit_last_d);
		hipFree(left_fit_last_d);
	}

}
